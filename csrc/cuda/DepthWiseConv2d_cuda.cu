#include "hip/hip_runtime.h"
// Copyright (c) Samson Wang. All Rights Reserved.
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>

#include <THC/THC.h>
#include <THC/THCAtomics.cuh>
#include <THC/THCDeviceUtils.cuh>

// For small input computation
template <typename T, int FixedKernelSize>
__global__ void DepthWiseConv2dSmallFForward(const T* bottom_data,
    const T* weight_data,
    const T* bias_data,
    const int channels, const int padding, const int height,
    const int width, const int in_kernel_size,
    const int out_height, const int out_width, const int batch_size, const bool forward,
    T* top_data) {
    const int in_num = height * width;
    const int out_num = out_height * out_width;
    const int pad_height = height + padding*2;
    const int pad_width = width + padding*2;
    const int pad_num = pad_height * pad_width;
    const int kernel_size = FixedKernelSize > 0 ? FixedKernelSize : in_kernel_size;
    const int kernel_num = kernel_size * kernel_size;
    const int thread_num = blockDim.x * blockDim.y;
    const int n_steps = blockDim.z * gridDim.z;
    const int out_num_total = n_steps * out_num;
    const int in_num_total = n_steps * in_num;

    const int tidz = threadIdx.z + blockDim.z * blockIdx.z;
    const int tidx = blockDim.x * threadIdx.y + threadIdx.x;
    __shared__ T w_shared[16*16];
    __shared__ T tmp_shared[4*16*16];

    // Initialize tmp shared for input data
    for (int off = threadIdx.z * thread_num + tidx; off < 4 * 256; off += thread_num) {
        tmp_shared[off] = T(0);
    }

        T bias = T(0);
//        if (bias_data != NULL) bias = bias_data[c];

    __syncthreads();
    const int bound = batch_size * channels;
    const int pidx = pad_width * (threadIdx.y + padding) + threadIdx.x + padding;
    const int opidx = pad_width * threadIdx.y + threadIdx.x;
    int tmp_p_off = threadIdx.z * pad_num;
    int tmp_w_off = threadIdx.z * kernel_num;
    int tmp_off = width * threadIdx.y + threadIdx.x + tidz * in_num;
    int tmp_out_off = threadIdx.y * out_width + threadIdx.x + tidz * out_num;
    int half_pad_off = pad_width * blockDim.y;
    int half_in_off = width * blockDim.y;
    int half_out_off = out_width * blockDim.y;
    for (int n_off = 0; n_off < bound; n_off += n_steps) {
        int n_z = n_off + tidz;
        int c = n_z % channels;
        int c_off = c * kernel_num;
        if (n_z < bound) {
        // Load kernels into shared memory
            for (int off = tidx; off < kernel_num; off += thread_num) {
                if (forward) {
                    w_shared[tmp_w_off + off] = weight_data[c_off + off];
                } else {
                    w_shared[tmp_w_off + off] = weight_data[c_off - off + kernel_num - 1];
                }
            }

        // Load input data input shared memory, pay attention to the padding.
            if (threadIdx.x < width && threadIdx.y < height) {
            tmp_shared[tmp_p_off + pidx] = bottom_data[tmp_off];
            if ((threadIdx.y + blockDim.y < height)) {
                tmp_shared[tmp_p_off + pidx + half_pad_off] = bottom_data[tmp_off + half_in_off];
            }
            }
        }

        __syncthreads();
/*
        if (threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0) {
            for (int i = 0; i < kernel_size; i++) {
                for (int j = 0; j < kernel_size; j++) {
                    printf("%f ", w_shared[i * kernel_size + j]);
                }
                printf("\n");
            }
            for (int i = 0; i < pad_height; i++) {
                for (int j = 0; j < pad_width; j++) {
                    printf("%f ", tmp_shared[i * pad_width + j]);
                }
                printf("\n");
            }
            printf("blockdim %d, %d, %d", blockDim.x, blockDim.y, blockDim.z);
        }
*/
        if (n_z < bound && threadIdx.x < out_width && threadIdx.y < out_height) {
            // To do the math
            T sum = T(0);
            T sum1 = T(0);
            int i_poff = tmp_p_off + opidx;
            #pragma unroll
            for (int i = 0; i < kernel_num; i+= kernel_size) {
                #pragma unroll
                for (int j = 0; j < kernel_size; j++) {
                    const T f = w_shared[i + tmp_w_off + j];
                    sum += tmp_shared[i_poff + j] * f;
                    if ((threadIdx.y + blockDim.y < out_height)) {
                        sum1 += tmp_shared[i_poff + j + half_pad_off] * f;
                    }
                }
                i_poff += pad_width;
            }
//            sum += bias;
            top_data[tmp_out_off] = sum;
            if ((threadIdx.y + blockDim.y < out_height)) {
                top_data[tmp_out_off + half_out_off] = sum1;
                //printf("top data %d, %d, %d, %d\n", threadIdx.x, threadIdx.y, tmp_out_off, half_out_off);
            }
        }
        tmp_off += in_num_total;
        tmp_out_off += out_num_total;
        __syncthreads();
    }
}

template <typename T>
__global__ void DepthWiseConv2dFForward(const T* bottom_data,
    const T* weight_data,
    const T* bias_data,
    const int channels, const int padding, const int height,
    const int width, const int kernel_size,
    const int out_height, const int out_width, const int output_size, const bool forward,
    T* top_data) {
    int tidx = threadIdx.y * blockDim.x + threadIdx.x;
    int o_idx = blockIdx.x * (blockDim.x - kernel_size + 1) + threadIdx.x;
    int o_idy = blockIdx.y * (blockDim.y - kernel_size + 1) + threadIdx.y;
    int c = (blockIdx.z) % channels;
    T bias = 0;
    if (bias_data != NULL) {
        bias = bias_data[c];
    }

    int kernel_num = kernel_size * kernel_size;
    __shared__ T w_shared[32];
    if (tidx < kernel_num) {
        if (forward) {
            w_shared[tidx] = weight_data[c * kernel_num + tidx];
        } else {
            w_shared[tidx] = weight_data[c * kernel_num + kernel_num - 1  - tidx];
        }
    }
    __syncthreads();

    __shared__ T tmp_shared[32*32];
for (int n_off = 0; n_off < output_size; n_off += gridDim.z) {
  if (blockIdx.z + n_off < output_size) {
    T sum = 0;
    //int n = blockIdx.z / channels;
//    int i_off_x = threadIdx.x - padding;
//    int i_off_y = threadIdx.y - padding;


    if (o_idx - padding >= 0 && o_idx - padding < width && o_idy - padding >=0 && o_idy - padding < height) {
        tmp_shared[threadIdx.y * blockDim.x + threadIdx.x] = bottom_data[(blockIdx.z + n_off) * width * height + (o_idy - padding) * width + o_idx - padding];
//        printf("tids %d, %d, oid %d, %d, padding %d, width %d, height %d, block %d, %d\n", tidx, tidy, o_idx, o_idy, padding, width, height, blockDim.x, blockDim.y);
    } else {
        tmp_shared[threadIdx.y * blockDim.x + threadIdx.x] = 0;
    }
    __syncthreads();
//    std::cout << tidx << " " << tidy << " " << " o " << o_idx << "  " << o_idy << " padding " << padding << " " << width << std::endl;
    if (o_idx >= 0 && o_idx < out_width && o_idy >=0 && o_idy < out_height && threadIdx.x < blockDim.x - kernel_size + 1 && threadIdx.y < blockDim.y - kernel_size + 1) {
        for (int i = 0; i < kernel_size; i++) {
            for (int j = 0; j < kernel_size; j++) {
                sum += tmp_shared[(threadIdx.y + i) * blockDim.x + threadIdx.x + j] * w_shared[i * kernel_size + j];
            }
        }
        top_data[(n_off + blockIdx.z) * out_width * out_height + (o_idy ) * out_width + o_idx ] = sum + bias;
    }
  } else {
//    printf("blockDim %d, %d, %d. gridDim %d, %d, %d os %d z %d off %d ch %d\n", blockDim.x, blockDim.y, blockDim.z, gridDim.x, gridDim.y, gridDim.z, output_size, blockIdx.z, n_off, channels);
  }
  __syncthreads();
}
}

template <typename T>
__global__ void DepthWiseConv2dLargeFForward(const T* bottom_data,
    const T* weight_data,
    const T* bias_data,
    const int channels, const int padding, const int height,
    const int width, const int kernel_size,
    const int out_height, const int out_width, const int batch_size,
    T* top_data) {
    __shared__ T true_r_shared[32*32];
    int n_idx = blockIdx.x * blockDim.y + threadIdx.y;
    const int n_num = gridDim.x * blockDim.y;

while (n_idx < channels * batch_size) {
    T* r_shared = true_r_shared;
    T sum[8] = {0};
    T tmp = 0;
    const int c = n_idx % channels;
    int valid_kernel_w = kernel_size;
    int valid_kernel_h = kernel_size;
    T* data = const_cast<T*> (bottom_data);
    data = data + n_idx * width * height;
    T* weight = const_cast<T*> (weight_data);
    weight = weight + c * kernel_size * kernel_size;
    const int y_shift = blockIdx.y - padding;
    const int x_shift = blockIdx.z - padding;
    if (blockIdx.y < padding) {
        valid_kernel_h = valid_kernel_h + y_shift;
        weight = weight - y_shift * kernel_size;
    } else if (blockIdx.y >= out_height - padding) {
        valid_kernel_h = valid_kernel_h - (blockIdx.y - out_height + padding + 1);
        data = data + y_shift * width;
    } else {
        data = data + y_shift * width;
    }
    if (blockIdx.z < padding) {
        valid_kernel_w = valid_kernel_w + x_shift;
        weight = weight - x_shift;
    } else if (blockIdx.z >= out_width - padding) {
        valid_kernel_w = valid_kernel_w - (blockIdx.z - out_width + padding + 1);
        data = data + x_shift;
    } else {
        data = data + x_shift;
    }

    const int y_num = (valid_kernel_h / 8) * 8;
    r_shared = r_shared + threadIdx.y * blockDim.x;
    for (int tidx = threadIdx.x; tidx < valid_kernel_w; tidx += blockDim.x) {
        int tmp_tidx_d = tidx;
        int tmp_tidx_w = tidx;
        for (int tidy = 0; tidy < y_num; tidy += 8) {
            #pragma unroll
            for (int j = 0; j < 8; j++) {
                sum[j] += data[j * width + tmp_tidx_d] * weight[j * kernel_size + tmp_tidx_w];
            }
            tmp_tidx_d = tmp_tidx_d + 8 * width;
            tmp_tidx_w = tmp_tidx_w + 8 * kernel_size;
        }
        for (int j = 0; j < valid_kernel_h - y_num; j++) {
            sum[j] += data[j * width + tmp_tidx_d] * weight[j * kernel_size + tmp_tidx_w];
        }
    }
    #pragma unroll
    for (int j = 0; j < 8; j++) {
        tmp += sum[j];
    }
    r_shared[threadIdx.x] = tmp;
    __syncthreads();
    if (threadIdx.x < 32) {
        for (int j = 32 + threadIdx.x; j < blockDim.x; j += 32) {
            tmp += r_shared[j];
        }
        r_shared[threadIdx.x] = tmp;
    }
    __syncthreads();
    if (threadIdx.x == 0) {
        tmp = r_shared[0];
        for (int j = 1; j < 32; j++) {
            tmp += r_shared[j];
        }
        top_data[n_idx * out_width * out_height + blockIdx.y * out_width + blockIdx.z] = tmp;
    }
    __syncthreads();
    n_idx += n_num;
}
}

at::Tensor DepthWiseConv2d_forward_cuda(const at::Tensor& input,
                                const at::Tensor& weight,
                                const at::Tensor& bias,
                                const int stride,
                                const int padding,
                                const int dilation,
                                const int groups) {
  AT_ASSERTM(input.type().is_cuda(), "input must be a CUDA tensor");

  auto batch_size = input.size(0);
  auto channels = input.size(1);
  auto height = input.size(2);
  auto width = input.size(3);

  auto kernel_size = weight.size(2);

  auto out_height = (height - kernel_size + 1 + padding * 2) / stride;
  auto out_width = (width - kernel_size + 1 + padding * 2) / stride;
  AT_ASSERTM(weight.size(0) == channels, "Weight output channel must be equal to Input channel");

  auto output = at::empty({batch_size, channels, out_height, out_width}, input.options());
  auto blockdim = 32;
  if (out_width < kernel_size && out_width + kernel_size - 1 < 32) {
    blockdim = kernel_size;
  } else if (out_width + kernel_size - 1 < 32) {
    blockdim = out_width + kernel_size - 1;
  }
  auto blocks_x = THCCeilDiv((long)out_width, blockdim-kernel_size+1L);
  auto blocks_y = THCCeilDiv((long)out_height, blockdim-kernel_size+1L);

  auto output_size = batch_size * channels;

  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  auto znum = output_size;
  if (znum > 2048) {
    znum = std::max((2048 / channels) * channels, channels);
  }
if (kernel_size > 16) {
    int blocks_x = kernel_size <= 1024 ? kernel_size : 1024;
    int blocks_y = (1024) / blocks_x;
    //dim3 grid((channels * batch_size + blocks_y - 1) / blocks_y, out_height, out_width);
    dim3 grid((channels * batch_size) / blocks_y / 2, out_height, out_width);
    dim3 block(blocks_x, blocks_y);
  AT_DISPATCH_FLOATING_TYPES(input.type(), "DepthWiseConv2d_forward", [&] {
    DepthWiseConv2dLargeFForward<scalar_t><<<grid, block, 0, stream>>>(
         input.contiguous().data<scalar_t>(),
         weight.contiguous().data<scalar_t>(),
         bias.contiguous().data<scalar_t>(),
         channels,
         padding,
         height,
         width,
         kernel_size,
         out_height,
         out_width,
         batch_size,
         output.data<scalar_t>());
  });
  THCudaCheck(hipGetLastError());

} else if (width + 2*padding > 16 || height + 2 * padding> 16) {
  dim3 grid(blocks_x, blocks_y, znum);
  dim3 block(blockdim, blockdim);

//  std::cout << "SHAPE dim x " << blocks_x << " dim y " << blocks_y << " nc " << batch_size * channels << std::endl;

//  std::cout << channels << " " << padding << " " << height << " " << width << " " << kernel_size << std::endl;
  //printf("blockdim %d, %d, %d, griddim %d, %d, %d outputsize %d\n", block.x, block.y, block.z, grid.x, grid.y, grid.z, output_size);

  //if (output.numel() == 0) {
  //  THCudaCheck(hipGetLastError());
  //  return output;
  //}
//niu
//  printf("blockdim %d, %d, %d, griddim %d, %d, %d outputsize %d, channels %d, width %d, height %d, padding %d, stride %d, bias %s, kernel_size %d\n", block.x, block.y, block.z, grid.x, grid.y, grid.z, batch_size, channels, width, height, padding, stride, bias.size(0), kernel_size);

  AT_DISPATCH_FLOATING_TYPES(input.type(), "DepthWiseConv2d_forward", [&] {
    DepthWiseConv2dFForward<scalar_t><<<grid, block, 0, stream>>>(
         input.contiguous().data<scalar_t>(),
         weight.contiguous().data<scalar_t>(),
         bias.contiguous().data<scalar_t>(),
         channels,
         padding,
         height,
         width,
         kernel_size,
         out_height,
         out_width,
         output_size,
         true,
         output.data<scalar_t>());
  });
  THCudaCheck(hipGetLastError());
} else {
  auto dimy = THCCeilDiv((long)(height > out_height ? height : out_height), 2L);
  auto blocks_x = 1;
  auto blocks_y = 1;
 
  dim3 grid(blocks_x, blocks_y, THCCeilDiv((long)channels*batch_size, 64L));
  dim3 block(width > out_width ? width : out_width, dimy, 8);
//  printf("Small blockdim %d, %d, %d, griddim %d, %d, %d outputsize %d, channels %d, width %d, height %d, padding %d, stride %d, bias %s, kernel_size %d\n", block.x, block.y, block.z, grid.x, grid.y, grid.z, batch_size, channels, width, height, padding, stride, bias.size(0), kernel_size);
if (kernel_size == 3) {
  AT_DISPATCH_FLOATING_TYPES(input.type(), "DepthWiseConv2dSmall_forward", [&] {
    DepthWiseConv2dSmallFForward<scalar_t, 3><<<grid, block, 0, stream>>>(
         input.contiguous().data<scalar_t>(),
         weight.contiguous().data<scalar_t>(),
         bias.contiguous().data<scalar_t>(),
         channels,
         padding,
         height,
         width,
         kernel_size,
         out_height,
         out_width,
         batch_size,
         true,
         output.data<scalar_t>());
  });
} else if (kernel_size == 5) {
  AT_DISPATCH_FLOATING_TYPES(input.type(), "DepthWiseConv2dSmall_forward", [&] {
    DepthWiseConv2dSmallFForward<scalar_t, 5><<<grid, block, 0, stream>>>(
         input.contiguous().data<scalar_t>(),
         weight.contiguous().data<scalar_t>(),
         bias.contiguous().data<scalar_t>(),
         channels,
         padding,
         height,
         width,
         kernel_size,
         out_height,
         out_width,
         batch_size,
         true,
         output.data<scalar_t>());
  });
} else {
  AT_DISPATCH_FLOATING_TYPES(input.type(), "DepthWiseConv2dSmall_forward", [&] {
    DepthWiseConv2dSmallFForward<scalar_t, 0><<<grid, block, 0, stream>>>(
         input.contiguous().data<scalar_t>(),
         weight.contiguous().data<scalar_t>(),
         bias.contiguous().data<scalar_t>(),
         channels,
         padding,
         height,
         width,
         kernel_size,
         out_height,
         out_width,
         batch_size,
         true,
         output.data<scalar_t>());
  });

}
  THCudaCheck(hipGetLastError());

}
  return output;
}

std::vector<at::Tensor> DepthWiseConv2d_backward_cuda(const at::Tensor& grad,
                                const at::Tensor& input,
                                const at::Tensor& weight,
                                const at::Tensor& bias,
                                const int stride,
                                const int padding,
                                const int dilation,
                                const int groups) {
  AT_ASSERTM(input.type().is_cuda(), "input must be a CUDA tensor");

  auto batch_size = input.size(0);
  auto channels = input.size(1);
  auto height = input.size(2);
  auto width = input.size(3);

  auto kernel_size = weight.size(2);

  auto out_height = (height - kernel_size + 1 + padding * 2) / stride;
  auto out_width = (width - kernel_size + 1 + padding * 2) / stride;
  AT_ASSERTM(weight.size(0) == channels, "Weight output channel must be equal to Input channel");

  // To deal with input grad computation.
  auto grad_input = at::empty({batch_size, channels, height, width}, grad.options());
  auto grad_weight = at::empty({channels, 1, kernel_size, kernel_size}, grad.options());
  auto grad_bias = at::empty({bias.size(0)}, grad.options());
  auto blockdim = 32;

  auto bwd_padding = kernel_size - 1 - padding;
  auto bwd_s = 1;
    std::cout << out_width << "x" << out_height << " Grad " << grad.size(2) << "x" << grad.size(3) << std::endl;
    std::cout << grad.size(3) - kernel_size + 1 + bwd_padding * 2 << " bwd " << bwd_padding << std::endl;
  AT_ASSERTM(width == (grad.size(3) - kernel_size + 1 + bwd_padding * 2), "grad_input computed size should be equal to input size")

  if (width < kernel_size && width + kernel_size - 1 < 32) {
    blockdim = kernel_size;
  } else if (width + kernel_size - 1 < 32) {
    blockdim = width + kernel_size - 1;
  }
  auto blocks_x = THCCeilDiv((long)width, blockdim-kernel_size+1L);
  auto blocks_y = THCCeilDiv((long)height, blockdim-kernel_size+1L);

  auto output_size = batch_size * channels;

  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  auto znum = output_size;
  if (znum > 2048) {
    znum = std::max((2048 / channels) * channels, channels);
  }
  if (out_width + 2*padding > 16 || out_height + 2 * padding> 16) {
    dim3 grid(blocks_x, blocks_y, znum);
    dim3 block(blockdim, blockdim);
  
    AT_DISPATCH_FLOATING_TYPES(input.type(), "DepthWiseConv2d_forward", [&] {
      DepthWiseConv2dFForward<scalar_t><<<grid, block, 0, stream>>>(
           grad.contiguous().data<scalar_t>(),
           weight.contiguous().data<scalar_t>(),
           bias.contiguous().data<scalar_t>(),
           channels,
           bwd_padding,
           out_height,
           out_width,
           kernel_size,
           height,
           width,
           output_size,
           false,
           grad_input.data<scalar_t>());
    });
    THCudaCheck(hipGetLastError());
  } else {
    auto dimy = THCCeilDiv((long)(out_height > height ? out_height : height), 2L);
    auto blocks_x = 1;
    auto blocks_y = 1;
 
    dim3 grid(blocks_x, blocks_y, THCCeilDiv((long)channels*batch_size, 64L));
    dim3 block(out_width > width ? out_width : width, dimy, 8);

    if (kernel_size == 3) {
      AT_DISPATCH_FLOATING_TYPES(input.type(), "DepthWiseConv2dSmall_forward", [&] {
        DepthWiseConv2dSmallFForward<scalar_t, 3><<<grid, block, 0, stream>>>(
             grad.contiguous().data<scalar_t>(),
             weight.contiguous().data<scalar_t>(),
             bias.contiguous().data<scalar_t>(),
             channels,
             bwd_padding,
             out_height,
             out_width,
             kernel_size,
             height,
             width,
             batch_size,
             false,
             grad_input.data<scalar_t>());
      });
        std::cout << "3 small" << std::endl;
    } else if (kernel_size == 5) {
      AT_DISPATCH_FLOATING_TYPES(input.type(), "DepthWiseConv2dSmall_forward", [&] {
        DepthWiseConv2dSmallFForward<scalar_t, 5><<<grid, block, 0, stream>>>(
             grad.contiguous().data<scalar_t>(),
             weight.contiguous().data<scalar_t>(),
             bias.contiguous().data<scalar_t>(),
             channels,
             bwd_padding,
             out_height,
             out_width,
             kernel_size,
             height,
             width,
             batch_size,
             false,
             grad_input.data<scalar_t>());
      });
        std::cout << "5 small" << std::endl;
    } else {
      AT_DISPATCH_FLOATING_TYPES(input.type(), "DepthWiseConv2dSmall_forward", [&] {
        DepthWiseConv2dSmallFForward<scalar_t, 0><<<grid, block, 0, stream>>>(
             grad.contiguous().data<scalar_t>(),
             weight.contiguous().data<scalar_t>(),
             bias.contiguous().data<scalar_t>(),
             channels,
             bwd_padding,
             out_height,
             out_width,
             kernel_size,
             height,
             width,
             batch_size,
             false,
             grad_input.data<scalar_t>());
      });
        std::cout << "Common small" << std::endl;
        //printf("<%d, %d, %d>\nGrid <%d, %d, %d>\nshape %d, %d, %d, %d\n", block.x, block.y, block.z, grid.x, grid.y, grid.z, width, height, out_width, out_height);
    }
    THCudaCheck(hipGetLastError());
  
  }

   //std::cout << "before return" << std::endl << out_width << std::endl << padding << std::endl << out_height << std::endl << width << std::endl;
   return std::vector<at::Tensor> {grad_input, grad_weight, grad_bias};
}

